
#include <hip/hip_runtime.h>
// convolve2d_kernel.cu

extern "C"
__global__ void convolve2DKernel(float* input, float* kernel, float* output, int width, int height, int kernelSize) {
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < height && j < width) {
		float sum = 0;
		for (int ki = 0; ki < kernelSize; ki++) {
			for (int kj = 0; kj < kernelSize; kj++) {
				int ii = i + ki - kernelSize / 2;
				int jj = j + kj - kernelSize / 2;
				if (ii >= 0 && ii < height && jj >= 0 && jj < width) {
					sum += input[ii * width + jj] * kernel[ki * kernelSize + kj];
				}
			}
		}
		output[i * width + j] = sum;
	}
}
