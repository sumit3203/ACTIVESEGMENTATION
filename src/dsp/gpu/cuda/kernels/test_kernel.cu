
#include <hip/hip_runtime.h>
﻿// test_kernel.cu

// -------------------------------
// This file contains two CUDA kernel functions:
// 1. helloKernel - prints a message from each GPU thread.
// 2. writeTest   - writes data into an output array from the GPU.
// These functions are compiled into a .ptx file and called from Java using JCuda.
// -------------------------------

// `extern "C"` prevents C++ name mangling so that JCuda (which uses C-style strings) can find the kernel function by name.
extern "C"

// This kernel will just print from each thread.
// Used to test if kernel is launching correctly.
__global__ void helloKernel() {
	printf("Hello from GPU thread %d\n", threadIdx.x);
}

extern "C"

// This kernel will write values to a GPU array,
// so we can copy it back in Java and confirm it worked.
__global__ void writeTest(int *output) {
	// Get the thread index in the current block.
	int tid = threadIdx.x;

	// Each thread writes its index * 10 into the output array.
	// Example: thread 0 → 0, thread 1 → 10, thread 2 → 20, etc.
	output[tid] = tid * 10;
}
