
#include <hip/hip_runtime.h>
/**
 * convolution_kernels.cu
 * Contains both 2D and 1D convolution kernels for image processing
 */

extern "C"
__global__ void convolve2DKernel(float* input, float* kernel, float* output,
	int width, int height, int kernelSize) {
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < height && j < width) {
		float sum = 0;
		int halfKernel = kernelSize / 2;

		for (int ki = 0; ki < kernelSize; ki++) {
			for (int kj = 0; kj < kernelSize; kj++) {
				int ii = i + ki - halfKernel;
				int jj = j + kj - halfKernel;

				// Boundary handling with clamp-to-edge
				if (ii >= 0 && ii < height && jj >= 0 && jj < width) {
					sum += input[ii * width + jj] * kernel[ki * kernelSize + kj];
				}
			}
		}
		output[i * width + j] = sum;
	}
}

extern "C"
__global__ void convolve1DKernel(float* input, float* kernel, float* output,
	int length, int kernelSize) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < length) {
		float sum = 0.0f;
		int halfKernel = kernelSize / 2;

		for (int k = -halfKernel; k <= halfKernel; k++) {
			int pos = idx + k;
			float value;

			// Handle boundaries with clamp-to-edge
			if (pos < 0) {
				value = input[0];
			}
			else if (pos >= length) {
				value = input[length - 1];
			}
			else {
				value = input[pos];
			}

			sum += value * kernel[k + halfKernel];
		}

		output[idx] = sum;
	}
}
